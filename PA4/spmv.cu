
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
//#include <cutil.h>
#define max(x,y) ((x)>(y)?(x):(y))
#define min(x,y) ((x)<(y)?)(x):(y))

extern int hipMemcpy();
extern int hipFree();
extern void _syncthreads();
extern int hipMemcpyToSymbol();
extern void MV_GPU_wrapper(int*, float*, float*, float*, int*, int, int, int);
extern int hipMalloc();
extern __global__ void mv_GPU(int, int, int, int*, float*, float*, float*, int*); 
//extern __shared__ float*;

int block_size = 0;
int grid_num = 0;
int threads_per_block = 0;
int max_non_zero_per_row = 0;

int compare(float* a, float* b, int size, double threshold)
{
	int i;
	for(i = 0; i < size; i++)
	{
		if(abs(a[i] - b[i]) > threshold) return 0;
	}
	return 1;
}

void normalMV(int nr, int* ptr, float* data, float* t, float* b, int* indices){
	int i, j;
	for(i = 0; i < nr; i++){
		for(j = ptr[i]; j < ptr[i + 1]; j++){
			t[i] = t[i] + data[j] * b[indices[j]];
		}
	}
}


extern void MV_GPU_wrapper(int* ptr, float* data, float* t, float* b, int* indices, int nr, int nc, int n){

	float* devO1Ptr;
	float* devI1Ptr;
	float* devI2Ptr;
	int* devIdPtr;
	int* devptr_ptr;
	
	hipMalloc((void**)&devO1Ptr, 4 * nr);
	hipMalloc((void**)&devI1Ptr, 4 * n);
	hipMalloc((void**)&devI2Ptr, 4 * nc);
	hipMemcpy(devO1Ptr, t, 4 * nr, hipMemcpyHostToDevice);
	hipMemcpy(devI1Ptr, data, 4 * n, hipMemcpyHostToDevice);
	hipMemcpy(devI2Ptr, b, 4 * nc, hipMemcpyHostToDevice);
	hipMalloc((void**)&devptr_ptr, 4 * (nr + 1));
	hipMalloc((void**)&devIdPtr, 4 * n);
	hipMemcpy(devptr_ptr, ptr, 4 * (nr + 1), hipMemcpyHostToDevice);
	hipMemcpy(devIdPtr, indices, 4 * n, hipMemcpyHostToDevice); 
	
	dim3 dimGrid(grid_num, 1);
	dim3 dimBlock(block_size, 1);
	dim3 threadsPerBlock(threads_per_block, 1);

//	printf("we have %d grids, %d blocks and %d threads per block\n", grid_num, block_size, threads_per_block);

	mv_GPU<<<dimGrid, dimBlock, threads_per_block>>>(nr, max_non_zero_per_row, block_size, devptr_ptr, devI1Ptr, devO1Ptr, devI2Ptr, devIdPtr);
	
	hipMemcpy(t, devO1Ptr, nr * 4, hipMemcpyDeviceToHost);

	hipFree(devO1Ptr);
	hipFree(devI1Ptr);
	hipFree(devI2Ptr);
	hipFree(devIdPtr);
	hipFree(devptr_ptr);

//data number of non zero
}


extern __global__ void mv_GPU(int nr, int mx, int blockSize, int* ptr, float* data, float* t, float* b, int* indices)
{
	int bx;
	int tx;
	float suif_tmp0;
//	__shared__ float _P1[];

//	__shared__ float* AS = _P1;
//	__shared__ float* BS = AS + (sizeof(float) * blockSize);

//	int blksz = blockSize;	
	int k, j;
	bx = blockIdx.x;
	tx = threadIdx.x;
	int ptr_cur;
	int ptr_next;
	if(tx <= -(blockSize * bx) + (nr - 1)){
//		suif_tmp0 = 0.0;
		suif_tmp0 = ((float* )(float(*)[])t)[tx + blockSize * bx];
		ptr_cur = ((int*)(int(*)[])ptr)[tx + blockSize * bx];
		ptr_next = ((int*)(int(*)[])ptr)[blockSize * bx + tx + 1];	
	}
	
//	for(k = 0; k < grid_num - 1; k++){
	//	if(tx <= -(block_size * k) + (nr - 1)){
	//		((float*)(float(*)[blksz])BS)[blksz * k + tx - blksz * k] = ((float*)(float(*)[])data)[blksz * k + tx];
	//	}
	
	//	__syncthreads();
		


	for(j = 0; j < mx; j++){
		
		if(tx <= -(blockSize * bx) + (nr - 1)){
			if(ptr_next > (ptr_cur + j)){
				//suif_tmp0 = suif_tmp0 + ((float*)(float(*)[blksz]BS)[(ptr_cur + j) - (blksz * k)] * b[indices[ptr_cur + j]];
				suif_tmp0 = suif_tmp0 + data[ptr_cur + j] * b[indices[ptr_cur + j]];
			}

		}
//	__syncthreads();

	}
	
	__syncthreads();
//}
	if(tx <= -(blockSize * bx) + (nr - 1)){
		((float*)(float(*)[])t)[tx + blockSize * bx] = suif_tmp0;	
	}		
}


int main(int argc, char** argv){

	FILE* fp;
	char line[1024];
	int* ptr, *indices;
	float *data, *b, *t_h, *t_d;
	int i, j;
	int n, nc, nr;
	

	if(argc < 2) abort();
	
	if((fp = fopen(argv[1], "r")) == NULL) abort();
	
	fgets(line, 128, fp);
	while(line[0] == '%'){
		fgets(line, 128, fp);
	}

	sscanf(line, "%d %d %d\n", &nr, &nc, &n);	
	ptr = (int*)malloc((nr + 1) * sizeof(int));
	indices = (int*)malloc(n * sizeof(int));
	data = (float*)malloc(n * sizeof(int));
	b = (float*)malloc(nc * sizeof(int));
	t_h = (float*)malloc(nr * sizeof(int));
	t_d = (float*)malloc(nr * sizeof(int));

	int lastr = 0;
	for(i = 0; i < n; i++)
	{
		int r;
		fscanf(fp, "%d %d %f\n", &r, &(indices[i]), &(data[i]));
		indices[i]--;
		if(r != lastr){
			ptr[r - 1] = i;
			lastr = r;
		}
		
	}
	ptr[nr] = n;
	int temp = 0;
	for(i = 0; i < nr; i++){
		temp = ptr[i + 1] - ptr[i];
		max_non_zero_per_row = max(temp, max_non_zero_per_row);
	}
	
	
	for(i = 0; i < nr; i++){
		t_h[i] = 0.0;
		t_d[i] = 0.0;
	}
	for(i = 0; i < nc; i++)
		b[i] = (float) rand() / 1111111111;

	fclose(fp);		

//	block_size = (nr + 31) / 32;
	block_size = sqrt(nr) + (sqrt(nr) / 2);
	grid_num = block_size / 2;	
	
	threads_per_block = 32;
		
	
	hipEvent_t start_event, end_event;
	float elapsed_time_seq, elapsed_time_gpu;
	hipEventCreate(&start_event);
	hipEventCreate(&end_event);
	hipEventRecord(start_event, 0);
	normalMV(nr, ptr, data, t_h, b, indices);
	hipEventRecord(end_event, 0);
	hipEventSynchronize(end_event);
	hipEventElapsedTime(&elapsed_time_seq, start_event, end_event);

	hipEventCreate(&start_event);
	hipEventCreate(&end_event);
	hipEventRecord(start_event, 0);
	MV_GPU_wrapper(ptr, data, t_d, b, indices, nr, nc, n);
//	cudaThreadSynchronize();
	hipEventRecord(end_event, 0);
	hipEventSynchronize(end_event);
	hipEventElapsedTime(&elapsed_time_gpu, start_event,  end_event);

	int res = compare(t_h, t_d, nr, 0.01);
	 
	if(res == 1)
		printf("VALID!\n Sequential Time: %.2f mesc\n  Parallel Time: %.2f mesc\n  Speedup = %.2f\n", elapsed_time_seq, elapsed_time_gpu, elapsed_time_seq / elapsed_time_gpu);
	else
		printf("INVALID...\n");
	
	return 0;
	
}
